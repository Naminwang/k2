/**
 * @brief Unittest for integral algorithm (this is really the application!)
 *
 * @copyright
 * Copyright (c)  2020  Xiaomi Corp.   (Author: Daniel Povey)
 *
 * @copyright
 * See LICENSE for clarification regarding multiple authors
 */

#include <gtest/gtest.h>

#include <string>

#include "k2/csrc/integral.h"

namespace k2 {

TEST(ComputeIntegral, DebugExclusiveSum) {
  ContextPtr c = GetCudaContext();
  Array1<char> keep(c, 16777216 + 1, (char)0);
  keep = keep.Range(0, keep.Dim() - 1);

  Array1<int32_t> sum(c, keep.Dim() + 1);
  ExclusiveSum(keep, &sum);
  K2_CHECK_EQ(sum.Back(), 0);
}

TEST(ComputeIntegral, SinglePointAtOrigin) {

  // configuration with a single unit mass located at the origin.
  // should be the default (zero initialization) anyway...
  Configuration configuration;
  InitConfigurationDefault(&configuration);
  configuration.masses[0] = 1.0;

  ContextPtr c = GetCudaContext();
  for (int32_t i = 1; i < 4; i++) {
    double r = 5.0 * i;  // cube radius (==half edge length)
    double integral_error;
    double integral = ComputeIntegral(c, configuration, r,
                                      1.0e-07,
                                      &integral_error);
    K2_LOG(INFO) << "For r = " << r << ", one mass at origin, integral = "
                 << integral << " with error " << integral_error;
  }
}

}  // namespace k2
